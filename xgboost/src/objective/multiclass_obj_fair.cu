#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2022 by XGBoost Contributors
 * \file multi_class_fair.cc
 * \brief Definition of multi-class fair classification objectives.
 * \author Authorname
 */
#include <dmlc/omp.h>

#include <vector>
#include <algorithm>
#include <limits>
#include <utility>

#include "xgboost/parameter.h"
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"
#include "xgboost/json.h"

#include "../common/common.h"
#include "../common/math.h"
#include "../common/transform.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(multiclass_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct SoftmaxMultiClassFairParam : public XGBoostParameter<SoftmaxMultiClassFairParam> {
  int num_class;
  // declare parameters
  DMLC_DECLARE_PARAMETER(SoftmaxMultiClassFairParam) {
    DMLC_DECLARE_FIELD(num_class).set_lower_bound(1)
        .describe("Number of output class in the multi-class classification.");
  }
};

class SoftmaxMultiClassFairObj : public ObjFunction {
 public:
  explicit SoftmaxMultiClassFairObj(bool output_prob)
  : output_prob_(output_prob) {}

  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kClassification; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    // Remove unused parameter compiler warning.
    (void) iter;
    if (info.labels.Size() == 0) {
      return;
    }
    CHECK(preds.Size() == (static_cast<size_t>(param_.num_class) * info.labels.Size()))
        << "SoftmaxMultiClassFairObj: label size and pred size does not match.\n"
        << "label.Size() * num_class: "
        << info.labels.Size() * static_cast<size_t>(param_.num_class) << "\n"
        << "num_class: " << param_.num_class << "\n"
        << "preds.Size(): " << preds.Size();
    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(preds.Size() / nclass);

    auto device = ctx_->gpu_id;
    out_gpair->SetDevice(device);
    info.labels.SetDevice(device);
    info.weights_.SetDevice(device);
    info.class_weight.SetDevice(device);
    info.last_pred.SetDevice(device);
    info.tau.SetDevice(device);
    preds.SetDevice(device);

    label_correct_.Resize(1);
    label_correct_.SetDevice(device);
    out_gpair->Resize(preds.Size());
    label_correct_.Fill(1);
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t idx,
                           common::Span<GradientPair> gpair,
                           common::Span<bst_float const> labels,
                           common::Span<bst_float const> preds,
                           common::Span<int> _label_correct,
                           common::Span<bst_float const> class_weight, 
                           common::Span<bst_float const> tau,
                           common::Span<bst_float const> last_pred,
                           common::Span<bst_float const> previous_pred,
                           common::Span<bst_float const> attribute_weight1,
                           common::Span<bst_float const> attribute_weight2) {
          common::Span<bst_float const> point = preds.subspan(idx * nclass, nclass);
          // Part of Softmax function
          bst_float wmax = std::numeric_limits<bst_float>::min();
          for (auto const i : point) { wmax = fmaxf(i, wmax); }
          double wsum = 0.0f;
          for (auto const i : point) { wsum += expf(i - wmax); }
          auto label = labels[idx];
          if (label < 0 || label >= nclass) {
            _label_correct[0] = 0;
            label = 0;
          }

          for (int k = 0; k < nclass; ++k) {
            // Computation duplicated to avoid creating a cache.
            bst_float p = expf(point[k] - wmax) / static_cast<float>(wsum);
            bst_float temp01 = attribute_weight1[idx] * p * class_weight[label];
            const bst_float h = temp01* (1.0f - p) + 1.0f/tau[0] + attribute_weight2[0];
            // idx-th data point
            p = temp01 + attribute_weight2[0] * (point[k] - previous_pred[idx * nclass + k]) + 1.0f/tau[0] * (point[k] - last_pred[idx * nclass + k]);
            p = label == k ? p - class_weight[k] * attribute_weight1[idx] : p;
            gpair[idx * nclass + k] = GradientPair(p, h);
          }
        }, common::Range{0, ndata}, ctx_->Threads(), device)
        .Eval(out_gpair, info.labels.Data(), &preds, &label_correct_, &info.class_weight, &info.tau, &info.last_pred,  &info.previous_pred, &info.attribute_weight1, &info.attribute_weight2);

    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag != 1) {
        LOG(FATAL) << "SoftmaxMultiClassFairObj: label must be in [0, num_class).";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float>* io_preds) const override {
    this->Transform(io_preds, output_prob_);
  }
  void EvalTransform(HostDeviceVector<bst_float>* io_preds) override {
    this->Transform(io_preds, true);
  }
  const char* DefaultEvalMetric() const override {
    return "mlogloss";
  }

  inline void Transform(HostDeviceVector<bst_float> *io_preds, bool prob) const {
    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(io_preds->Size() / nclass);

    auto device = io_preds->DeviceIdx();
    if (prob) {
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
            common::Span<bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            common::Softmax(point.begin(), point.end());
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds);
    } else {
      io_preds->SetDevice(device);
      HostDeviceVector<bst_float> max_preds;
      max_preds.SetDevice(device);
      max_preds.Resize(ndata);
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<const bst_float> _preds,
                             common::Span<bst_float> _max_preds) {
            common::Span<const bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            _max_preds[_idx] =
                common::FindMaxIndex(point.cbegin(), point.cend()) -
                point.cbegin();
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds, &max_preds);
      io_preds->Resize(max_preds.Size());
      io_preds->Copy(max_preds);
    }
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    if (this->output_prob_) {
      out["name"] = String("multi:softprob_fair");
    } else {
      out["name"] = String("multi:softmax_fair");
    }
    out["softmax_fair_multiclass_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["softmax_fair_multiclass_param"], &param_);
  }

 private:
  // output probability
  bool output_prob_;
  // parameter
  SoftmaxMultiClassFairParam param_;
  // Cache for max_preds
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(SoftmaxMultiClassFairParam);

XGBOOST_REGISTER_OBJECTIVE(SoftmaxMultiClass, "multi:softmax_fair")
.describe("Softmax for multi-class classification, output class index.")
.set_body([]() { return new SoftmaxMultiClassFairObj(false); });

XGBOOST_REGISTER_OBJECTIVE(SoftprobMultiClass, "multi:softprob_fair")
.describe("Softmax for multi-class classification, output probability distribution.")
.set_body([]() { return new SoftmaxMultiClassFairObj(true); });

}  // namespace obj
}  // namespace xgboost
