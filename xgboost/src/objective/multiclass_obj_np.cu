#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2022 by XGBoost Contributors
 * \file multi_class_np.cu
 * \brief Definition of multi-class Neyman Pearson classification objectives.
 * \author Authorname
 */
#include <dmlc/omp.h>

#include <vector>
#include <algorithm>
#include <limits>
#include <utility>

#include "xgboost/parameter.h"
#include "xgboost/data.h"
#include "xgboost/logging.h"
#include "xgboost/objective.h"
#include "xgboost/json.h"

#include "../common/common.h"
#include "../common/math.h"
#include "../common/transform.h"

namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(multiclass_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct SoftmaxMultiClassNPParam : public XGBoostParameter<SoftmaxMultiClassNPParam> {
  int num_class;
  // declare parameters
  DMLC_DECLARE_PARAMETER(SoftmaxMultiClassNPParam) {
    DMLC_DECLARE_FIELD(num_class).set_lower_bound(1)
        .describe("Number of output class in the multi-class classification.");
  }
};

class SoftmaxMultiClassNPObj : public ObjFunction {
 public:
  explicit SoftmaxMultiClassNPObj(bool output_prob)
  : output_prob_(output_prob) {}

  void Configure(Args const& args) override {
    param_.UpdateAllowUnknown(args);
  }

  ObjInfo Task() const override { return ObjInfo::kClassification; }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    // Remove unused parameter compiler warning.
    (void) iter;
    if (info.labels.Size() == 0) {
      return;
    }
    CHECK(preds.Size() == (static_cast<size_t>(param_.num_class) * info.labels.Size()))
        << "SoftmaxMultiClassNPObj: label size and pred size does not match.\n"
        << "label.Size() * num_class: "
        << info.labels.Size() * static_cast<size_t>(param_.num_class) << "\n"
        << "num_class: " << param_.num_class << "\n"
        << "preds.Size(): " << preds.Size();
    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(preds.Size() / nclass);

    auto device = ctx_->gpu_id;
    out_gpair->SetDevice(device);
    info.labels.SetDevice(device);
    // info.weights_.SetDevice(device);
    info.class_weight.SetDevice(device);
    info.last_pred.SetDevice(device);
    info.tau.SetDevice(device);
    preds.SetDevice(device);

    label_correct_.Resize(1);
    label_correct_.SetDevice(device);
    out_gpair->Resize(preds.Size());
    label_correct_.Fill(1);
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t idx,
                           common::Span<GradientPair> gpair,
                           common::Span<bst_float const> labels,
                           common::Span<bst_float const> preds,
                           common::Span<int> _label_correct,
                           common::Span<bst_float const> class_weight, 
                           common::Span<bst_float const> tau,
                           common::Span<bst_float const> last_pred) {
          common::Span<bst_float const> point = preds.subspan(idx * nclass, nclass);
          // Part of Softmax function
          bst_float wmax = std::numeric_limits<bst_float>::min();
          for (auto const i : point) { wmax = fmaxf(i, wmax); }
          double wsum = 0.0f;
          for (auto const i : point) { wsum += expf(i - wmax); }
          auto label = labels[idx];
          if (label < 0 || label >= nclass) {
            _label_correct[0] = 0;
            label = 0;
          }

          for (int k = 0; k < nclass; ++k) {
            // Computation duplicated to avoid creating a cache.
            // class_weight is `lambda + w_k`
            bst_float p = expf(point[k] - wmax) / static_cast<float>(wsum);
            const bst_float h = p * (1.0f - p) * class_weight[label] + 1.0f/tau[0];
            p = p * class_weight[label] + 1.0f/tau[0] * (point[k] - last_pred[idx * nclass + k]);
            p = label == k ? p - class_weight[k] : p;
            gpair[idx * nclass + k] = GradientPair(p, 2 * h);
          }
        }, common::Range{0, ndata}, ctx_->Threads(), device)
        .Eval(out_gpair, info.labels.Data(), &preds, &label_correct_, &info.class_weight, &info.tau, &info.last_pred);

    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag != 1) {
        LOG(FATAL) << "SoftmaxMultiClassNPObj: label must be in [0, num_class).";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float>* io_preds) const override {
    this->Transform(io_preds, output_prob_);
  }
  void EvalTransform(HostDeviceVector<bst_float>* io_preds) override {
    this->Transform(io_preds, true);
  }
  const char* DefaultEvalMetric() const override {
    return "mlogloss";
  }

  inline void Transform(HostDeviceVector<bst_float> *io_preds, bool prob) const {
    const int nclass = param_.num_class;
    const auto ndata = static_cast<int64_t>(io_preds->Size() / nclass);

    auto device = io_preds->DeviceIdx();
    if (prob) {
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
            common::Span<bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            common::Softmax(point.begin(), point.end());
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds);
    } else {
      io_preds->SetDevice(device);
      HostDeviceVector<bst_float> max_preds;
      max_preds.SetDevice(device);
      max_preds.Resize(ndata);
      common::Transform<>::Init(
          [=] XGBOOST_DEVICE(size_t _idx, common::Span<const bst_float> _preds,
                             common::Span<bst_float> _max_preds) {
            common::Span<const bst_float> point =
                _preds.subspan(_idx * nclass, nclass);
            _max_preds[_idx] =
                common::FindMaxIndex(point.cbegin(), point.cend()) -
                point.cbegin();
          },
          common::Range{0, ndata}, this->ctx_->Threads(), device)
          .Eval(io_preds, &max_preds);
      io_preds->Resize(max_preds.Size());
      io_preds->Copy(max_preds);
    }
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    if (this->output_prob_) {
      out["name"] = String("multi:softprob_np");
    } else {
      out["name"] = String("multi:softmax_np");
    }
    out["softmax_np_multiclass_param"] = ToJson(param_);
  }

  void LoadConfig(Json const& in) override {
    FromJson(in["softmax_np_multiclass_param"], &param_);
  }

 private:
  // output probability
  bool output_prob_;
  // parameter
  SoftmaxMultiClassNPParam param_;
  // Cache for max_preds
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(SoftmaxMultiClassNPParam);

XGBOOST_REGISTER_OBJECTIVE(SoftmaxMultiClassNP, "multi:softmax_np")
.describe("Softmax for multi-class classification, output class index.")
.set_body([]() { return new SoftmaxMultiClassNPObj(false); });

XGBOOST_REGISTER_OBJECTIVE(SoftprobNPMultiClassNP, "multi:softprob_np")
.describe("Softmax for multi-class classification, output probability distribution.")
.set_body([]() { return new SoftmaxMultiClassNPObj(true); });
}  // namespace obj
}  // namespace xgboost
